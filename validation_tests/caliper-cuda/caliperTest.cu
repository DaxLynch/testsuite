#include "hip/hip_runtime.h"
// Copyright (c) 2015-2020, Lawrence Livermore National Security, LLC.
// See top-level LICENSE file for details.

// A C++ Caliper instrumentation and ConfigManager example

//   Usage: $ cxx-example [-P <configuration-string>] <iterations>
// For example, "$ cxx-example -P runtime-report" will print a
// hierarchical runtime summary for all annotated regions.

#include <caliper/cali.h>
#include <caliper/cali-manager.h>

#include "hip/hip_runtime.h"

#include <time.h>

#include <cstring>
#include <iostream>
#include <string>

#define N 10

void print_help(const cali::ConfigManager& mgr)
{
    std::cerr << "Usage: cxx-example [-P caliper-config(arg=...,),...] [iterations]."
              << "\nAvailable configurations: ";

    auto configs = mgr.available_config_specs();

    // Print info on all available ConfigManager configurations.
    for (auto str : configs)
        std::cerr << "\n" << mgr.get_documentation_for_spec(str.c_str());

    std::cerr << std::endl;
}

__global__ void add(int *a, int *b, int *c)
{
    int tID = blockIdx.x;
    if (tID < N)
    {
        c[tID] = a[tID] + b[tID];
    }
}

double foo(int i)
{
    //   A function annotation. Opens region "function=foo" in Caliper,
    // and automatically closes it at the end of the function.
    CALI_CXX_MARK_FUNCTION;

    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;

    hipMalloc((void **) &dev_a, N*sizeof(int));
    hipMalloc((void **) &dev_b, N*sizeof(int));
    hipMalloc((void **) &dev_c, N*sizeof(int));

    // Fill Arrays
    for (int i = 0; i < N; i++)
    {
        a[i] = i,
        b[i] = i;
    }
    hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);

    add<<<N,1>>>(dev_a, dev_b, dev_c);

    hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);

    return 0.5*i;
}

int main(int argc, char* argv[])
{
    //   The ConfigManager manages built-in or custom Caliper measurement
    // configurations, and provides an API to control performance profiling.
    cali::ConfigManager mgr;

    //   Parse command-line arguments. Let users choose a Caliper performance
    // profiling configuration via the "-P" command-line argument.
    std::string configstr;
    int iterations = 4;

    for (int a = 1; a < argc; ++a) {
        if (strcmp(argv[a], "-h") == 0 || strcmp(argv[a], "--help") == 0) {
            print_help(mgr);
            return 0;
        } else if (strcmp(argv[a], "-P") == 0) {
            ++a;
            if (argc > a)
                configstr = argv[a];
            else {
                std::cerr << "Expected config string after \"-P\"";
                return 1;
            }
        } else {
            try {
                iterations = std::stoi(argv[a]);
            } catch (std::invalid_argument) {
                std::cerr << "Invalid argument: \"" << argv[a]
                          << "\". Expected a number."
                          << std::endl;
                return 2;
            }
        }
    }

    //   Enable the requested performance measurement channels and start
    // profiling.
    mgr.add(configstr.c_str());

    if (mgr.error())
        std::cerr << "Caliper config error: " << mgr.error_msg() << std::endl;

    mgr.start();

    //   Add some run metadata information to be stored in the
    // performance profiles.
    cali_set_global_int_byname("iterations", iterations);
    cali_set_global_string_byname("caliper.config", configstr.c_str());

    //   Mark begin of the current function. Must be manually closed.
    // Opens region "function=main" in Caliper.
    CALI_MARK_FUNCTION_BEGIN;

    // Mark a code region. Opens region "annotation=init" in Caliper.
    CALI_MARK_BEGIN("init");
    double t = 0;
    CALI_MARK_END("init");

    // Mark a loop. Opens region "loop=mainloop" in Caliper.
    CALI_CXX_MARK_LOOP_BEGIN(loop_ann, "mainloop");

    for (int i = 0; i < iterations; ++i) {
        //   Mark loop iterations of an annotated loop.
        // Sets "iteration#main loop=<i> in Caliper.
        CALI_CXX_MARK_LOOP_ITERATION(loop_ann, i);

        //   A Caliper snapshot taken at this point will contain
        // { "function"="main", "loop"="mainloop", "iteration#main loop"=<i> }

        t += foo(i);
    }

    // Mark the end of the "loop=mainloop" region.
    CALI_CXX_MARK_LOOP_END(loop_ann);
    // Mark the end of the "function=main" region.
    CALI_MARK_FUNCTION_END;

    //   Trigger output in all Caliper control channels.
    // This should be done after all measurement regions have been closed.
    mgr.flush();
}
