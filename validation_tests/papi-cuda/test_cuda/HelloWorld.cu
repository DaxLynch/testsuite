/****************************/
/* THIS IS OPEN SOURCE CODE */
/****************************/

/**
 * @file    HelloWorld.cu
 * @author  Heike Jagode
 *          jagode@eecs.utk.edu
 * Mods:	Anustuv Pal
 *			anustuv@icl.utk.edu
 * Mods:	<your name here>
 *			<your email address>
 * test case for Example component
 *
 *
 * @brief
 *  This file is a very simple HelloWorld C example which serves (together
 *	with its Makefile) as a guideline on how to add tests to components.
 *  The papi configure and papi Makefile will take care of the compilation
 *	of the component tests (if all tests are added to a directory named
 *	'tests' in the specific component dir).
 *	See components/README for more details.
 *
 *	The string "Hello World!" is mangled and then restored.
 *
 *  CUDA Context notes for CUPTI_11: Although a cudaSetDevice() will create a
 *  primary context for the device that allows kernel execution; PAPI cannot
 *  use a primary context to control the Nvidia Performance Profiler.
 *  Applications must create a context using cuCtxCreate() that will execute
 *  the kernel, this must be done prior to the PAPI_add_events() invocation in
 *  the code below. If multiple GPUs are in use, each requires its own context,
 *  and that context should be active when PAPI_events are added for each
 *  device.  Which means using Seperate PAPI_add_events() for each device. For
 *  an example see simpleMultiGPU.cu.
 *
 *  There are three points below where cuCtxCreate() is called, this code works
 *  if any one of them is used alone.
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#ifdef PAPI
#include "papi.h"
#endif

#define STEP_BY_STEP_DEBUG 0 /* helps debug CUcontext issues. */

// Prototypes
__global__ void helloWorld(char*);


// Host function
int main(int argc, char** argv)
{
    hipCtx_t getCtx=NULL, sessionCtx=NULL;
    hipError_t hipError_t;
    hipError_t cuError; (void) cuError;
    int retval = 0;
#ifdef PAPI

    /* PAPI Initialization */
    int papi_errno = PAPI_library_init( PAPI_VER_CURRENT );
    if( papi_errno != PAPI_VER_CURRENT ) {
        fprintf(stderr, "PAPI_library_init failed.\n");
        retval++;
    }

    printf( "PAPI_VERSION     : %4d %6d %7d\n",
        PAPI_VERSION_MAJOR( PAPI_VERSION ),
        PAPI_VERSION_MINOR( PAPI_VERSION ),
        PAPI_VERSION_REVISION( PAPI_VERSION ) );

    int i;
    int EventSet = PAPI_NULL;
    int eventCount = argc - 1;

    /* if no events passed at command line, just report test skipped. */
    if (eventCount == 0) {
        fprintf(stderr, "No eventnames specified at command line.\n");
        retval++;
    }

    long long *values = (long long *) calloc(eventCount, sizeof (long long));
    if (values == NULL) {
        fprintf(stderr, "Failed to allocate memory for values.\n");
        retval++;
    }
    int *events = (int *) calloc(eventCount, sizeof (int));
    if (events == NULL) {
        fprintf(stderr, "Failed to allocate memory for events.\n");
        retval++;
    }
    /* convert PAPI native events to PAPI code */
    for( i = 0; i < eventCount; i++ ){
        papi_errno = PAPI_event_name_to_code( argv[i+1], &events[i] );
        if( papi_errno != PAPI_OK ) {
            fprintf(stderr, "Check event name: %s", argv[i+1] );
        }
        printf("Name %s --- Code: %#x\n", argv[i+1], events[i]);
    }

    if (STEP_BY_STEP_DEBUG) {
        cuCtxGetCurrent(&getCtx);
        fprintf(stderr, "%s:%s:%i before PAPI_create_eventset() getCtx=%p.\n", __FILE__, __func__, __LINE__, getCtx);
    }

    papi_errno = PAPI_create_eventset( &EventSet );
    if( papi_errno != PAPI_OK ) {
        fprintf(stderr, "Cannot create eventset.\n");
        retval++;
    }

    if (STEP_BY_STEP_DEBUG) {
        cuCtxGetCurrent(&getCtx);
        fprintf(stderr, "%s:%s:%i before PAPI_add_events(), getCtx=%p.\n", __FILE__, __func__, __LINE__, getCtx);
    }

    // If multiple GPUs/contexts were being used, you'd need to
    // create contexts for each device. See, for example,
    // simpleMultiGPU.cu.

    // Context Create. We will use this one to run our kernel.
    cuError = cuCtxCreate(&sessionCtx, 0, 0); // Create a context, NULL flags, Device 0.
    if (cuError != CUDA_SUCCESS) {
        fprintf(stderr, "Failed to create cuContext.\n");
        retval++;
    }

    if (STEP_BY_STEP_DEBUG) {
        cuCtxGetCurrent(&getCtx);
        fprintf(stderr, "%s:%s:%i after cuCtxCreate(&sessionCtx), about to PAPI_start(), sessionCtx=%p, getCtx=%p.\n", __FILE__, __func__, __LINE__, sessionCtx, getCtx);
    }

    papi_errno = PAPI_add_events( EventSet, events, eventCount );
    if (papi_errno == PAPI_ENOEVNT) {
        fprintf(stderr, "Event name does not exist for component.");
    }
    if( papi_errno != PAPI_OK ) {
        fprintf(stderr, "PAPI_add_events failed.\n");
        retval++;
    }

    if (STEP_BY_STEP_DEBUG) {
        cuCtxGetCurrent(&getCtx);
        fprintf(stderr, "%s:%s:%i before PAPI_start(), getCtx=%p.\n", __FILE__, __func__, __LINE__, getCtx);
    }

    papi_errno = PAPI_start( EventSet );
    if( papi_errno != PAPI_OK ) {
        fprintf(stderr, "PAPI_start failed.\n");
        retval++;
    }

    if (STEP_BY_STEP_DEBUG) {
        cuCtxGetCurrent(&getCtx);
        fprintf(stderr, "%s:%s:%i after PAPI_start(), getCtx=%p.\n", __FILE__, __func__, __LINE__, getCtx);
    }

#endif

    int j;

    // desired output
    char str[] = "Hello World!";

    // mangle contents of output
    // the null character is left intact for simplicity
    for(j = 0; j < 12; j++) {
        str[j] -= j;
    }

    printf("mangled str=%s\n", str);

    // allocate memory on the device
    char *d_str;
    size_t size = sizeof(str);
    hipMalloc((void**)&d_str, size);

    if (STEP_BY_STEP_DEBUG) {
        hipCtxGetCurrent(&getCtx);
        fprintf(stderr, "%s:%s:%i after cudaMalloc() getCtx=%p.\n", __FILE__, __func__, __LINE__, getCtx);
    }

    // copy the string to the device
    hipMemcpy(d_str, str, size, hipMemcpyHostToDevice);

    if (STEP_BY_STEP_DEBUG) {
        hipCtxGetCurrent(&getCtx);
        fprintf(stderr, "%s:%s:%i after cudaMemcpy(ToDevice) getCtx=%p.\n", __FILE__, __func__, __LINE__, getCtx);
    }

    // set the grid and block sizes
    dim3 dimGrid(2); // one block per word
    dim3 dimBlock(6); // one thread per character

    // invoke the kernel
    helloWorld<<< dimGrid, dimBlock >>>(d_str);

    hipError_t = hipGetLastError();
    if (STEP_BY_STEP_DEBUG) {
        fprintf(stderr, "%s:%s:%i Kernel Return Code: %s.\n", __FILE__, __func__, __LINE__, hipGetErrorString(hipError_t));
    }

    if (STEP_BY_STEP_DEBUG) {
        hipCtxGetCurrent(&getCtx);
        fprintf(stderr, "%s:%s:%i After Kernel Execution: getCtx=%p.\n", __FILE__, __func__, __LINE__, getCtx);
    }

    // retrieve the results from the device
    hipMemcpy(str, d_str, size, hipMemcpyDeviceToHost);

    if (STEP_BY_STEP_DEBUG) {
        hipCtxGetCurrent(&getCtx);
        fprintf(stderr, "%s:%s:%i after cudaMemcpy(ToHost) getCtx=%p.\n", __FILE__, __func__, __LINE__, getCtx);
    }

    // free up the allocated memory on the device
    hipFree(d_str);

    if (STEP_BY_STEP_DEBUG) {
        hipCtxGetCurrent(&getCtx);
        fprintf(stderr, "%s:%s:%i after cudaFree() getCtx=%p.\n", __FILE__, __func__, __LINE__, getCtx);
    }


#ifdef PAPI
    papi_errno = PAPI_read( EventSet, values );
    if( papi_errno != PAPI_OK ) {
        fprintf(stderr, "PAPI_read failed.\n");
        retval++;
    }

    if (STEP_BY_STEP_DEBUG) {
        cuCtxGetCurrent(&getCtx);
        fprintf(stderr, "%s:%s:%i after PAPI_read getCtx=%p.\n", __FILE__, __func__, __LINE__, getCtx);
    }

    for( i = 0; i < eventCount; i++ ) {
        printf("read: %12lld \t=0X%016llX \t\t --> %s \n", values[i], values[i], argv[i+1]);
    }

    papi_errno = cuCtxPopCurrent(&getCtx);
    if( papi_errno != CUDA_SUCCESS) {
        fprintf( stderr, "cuCtxPopCurrent failed, papi_errno=%d (%s)\n", papi_errno, PAPI_strerror(papi_errno) );
        retval++;
    }

    if (STEP_BY_STEP_DEBUG) {
        cuCtxGetCurrent(&getCtx);
        fprintf(stderr, "%s:%s:%i after cuCtxPopCurrent() getCtx=%p.\n", __FILE__, __func__, __LINE__, getCtx);
    }

    papi_errno = PAPI_stop( EventSet, values );
    if( papi_errno != PAPI_OK ) {
        fprintf(stderr, "PAPI_stop failed.\n");
        retval++;
    }

    if (STEP_BY_STEP_DEBUG) {
        cuCtxGetCurrent(&getCtx);
        fprintf(stderr, "%s:%s:%i after PAPI_stop getCtx=%p.\n", __FILE__, __func__, __LINE__, getCtx);
    }

    papi_errno = PAPI_cleanup_eventset(EventSet);
    if( papi_errno != PAPI_OK ) {
        fprintf(stderr, "PAPI_cleanup_eventset failed.\n");
        retval++;
    }

    if (STEP_BY_STEP_DEBUG) {
        cuCtxGetCurrent(&getCtx);
        fprintf(stderr, "%s:%s:%i after PAPI_cleanup_eventset getCtx=%p.\n", __FILE__, __func__, __LINE__, getCtx);
    }

    papi_errno = PAPI_destroy_eventset(&EventSet);
    if (papi_errno != PAPI_OK) {
        fprintf(stderr, "PAPI_destroy_eventset failed.\n");
        retval++;
    }

    if (STEP_BY_STEP_DEBUG) {
        cuCtxGetCurrent(&getCtx);
        fprintf(stderr, "%s:%s:%i after PAPI_destroy_eventset getCtx=%p.\n", __FILE__, __func__, __LINE__, getCtx);
    }


    for( i = 0; i < eventCount; i++ ) {
        printf("stop: %12lld \t=0X%016llX \t\t --> %s \n", values[i], values[i], argv[i+1]);
    }
#endif

    if (STEP_BY_STEP_DEBUG) {
        fprintf(stderr, "%s:%s:%i before hipCtxDestroy sessionCtx=%p.\n", __FILE__, __func__, __LINE__, sessionCtx);
    }

    // Test destroying the session Context.
    if (sessionCtx != NULL) {
        hipCtxDestroy(sessionCtx);
    }

    if (STEP_BY_STEP_DEBUG) {
        hipCtxGetCurrent(&getCtx);
        fprintf(stderr, "%s:%s:%i after cuCtxDestroy(%p) getCtx=%p.\n", __FILE__, __func__, __LINE__, sessionCtx, getCtx);
    }

#ifdef PAPI
    PAPI_shutdown();

    if (STEP_BY_STEP_DEBUG) {
        cuCtxGetCurrent(&getCtx);
        fprintf(stderr, "%s:%s:%i after PAPI_shutdown getCtx=%p.\n", __FILE__, __func__, __LINE__, getCtx);
    }

#endif
    return retval;
}


// Device kernel
__global__ void
helloWorld(char* str)
{
    // determine where in the thread grid we are
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // unmangle output
    str[idx] += idx;
}
